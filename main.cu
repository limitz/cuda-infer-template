#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <math.h>
#include <sys/stat.h>
#if USE_NVJPEG
#include <nvjpeg.h>
#else
#include <jpeglib.h>
#endif

#include <kinect.h>
#include <display.h>
#include <pthread.h>
#include <math.h>
#include <inference.h>
#include <operators.h>
#include <asyncwork.h>
#include <jpegcodec.h>
#include <file.h>

#ifndef TITLE
#define TITLE "CUDA INFERENCE DEMO"
#endif

#ifndef USE_NVJPEG
#define USE_NVJPEG 0
#endif

//width and height defines come from inference.h at the moment

static uint8_t* imageBuffer = {0};

__global__
void f_test(float4* out, int pitch_out, int width, int height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	out[y * pitch_out / sizeof(float4) + x] = make_float4(
			(float) x / width, 
			(float) y / height, 
			0, 1);
}

// RGB interleaved as 3 byte tupels
__global__
void f_jpeg(float4* out, int pitch_out, uint8_t* rgb, int width, int height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	out[y * pitch_out / sizeof(float4) + x] = make_float4(
			rgb[0 + y * width * 3 + x * 3] / 255.0f,
			rgb[1 + y * width * 3 + x * 3] / 255.0f,
			rgb[2 + y * width * 3 + x * 3] / 255.0f,
			1);
}
__global__
void f_normalize(float* normalized, uint8_t* rgb, size_t width, size_t height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;
	size_t scstride = (width/SCALE) * (height/SCALE);
	size_t offset = y * width + x;
	size_t soffset = (y / SCALE) * (width/SCALE) + x / SCALE;

	normalized[soffset + 0 * scstride] = (rgb[offset*3 + 0]/255.0f - 0.485f) / (0.229f); 
	normalized[soffset + 1 * scstride] = (rgb[offset*3 + 1]/255.0f - 0.456f) / (0.224f); 
	normalized[soffset + 2 * scstride] = (rgb[offset*3 + 2]/255.0f - 0.406f) / (0.225f); 
}

__global__
void f_segment(float4* out, int pitch_out, int* seg, int width, int height)
{
	int x = (blockIdx.x * blockDim.x + threadIdx.x);
	int y = (blockIdx.y * blockDim.y + threadIdx.y);
	if (x >= width || y >= height) return;

	float alpha = 0.4;
	int classification = seg[(y/SCALE) * (width/SCALE) + (x/SCALE)];
	float4 color = classification ? make_float4(
			alpha/2 + alpha/2 * __sinf((classification/20.0f+0.00f) * 2 * M_PI),
			alpha/2 + alpha/2 * __sinf((classification/20.0f+0.33f) * 2 * M_PI),
			alpha/2 + alpha/2 * __sinf((classification/20.0f+0.66f) * 2 * M_PI),
			alpha) : make_float4(0,0,0,0);

	int idx = y * pitch_out/sizeof(float4) + x;
	out[idx] = out[idx] * (1-color.w) + color;
}

int smToCores(int major, int minor)
{
	switch ((major << 4) | minor)
	{
		case (9999 << 4 | 9999):
			return 1;
		case 0x30:
		case 0x32:
		case 0x35:
		case 0x37:
			return 192;
		case 0x50:
		case 0x52:
		case 0x53:
			return 128;
		case 0x60:
			return 64;
		case 0x61:
		case 0x62:
			return 128;
		case 0x70:
		case 0x72:
		case 0x75:
			return 64;
		case 0x80:
		case 0x86:
			return 64;
		default:
			return 0;
	};
}

void selectGPU()
{
	int rc;
	int maxId = -1;
	uint16_t maxScore = 0;
	int count = 0;
	hipDeviceProp_t prop;

	rc = hipGetDeviceCount(&count);
	if (hipSuccess != rc) throw "hipGetDeviceCount error";
	if (count == 0) throw "No suitable cuda device found";

	for (int id = 0; id < count; id++)
	{
		rc = hipGetDeviceProperties(&prop, id);
		if (hipSuccess != rc) throw "Unable to get device properties";
		if (prop.computeMode == hipComputeModeProhibited) 
		{
			printf("GPU %d: PROHIBITED\n", id);
			continue;
		}
		int sm_per_multiproc = smToCores(prop.major, prop.minor);
		
		printf("GPU %d: \"%s\"\n", id, prop.name);
		printf(" - Compute capability: %d.%d\n", prop.major, prop.minor);
		printf(" - Multiprocessors:    %d\n", prop.multiProcessorCount);
		printf(" - SMs per processor:  %d\n", sm_per_multiproc);
		printf(" - Clock rate:         %d\n", prop.clockRate);

		uint64_t score =(uint64_t) prop.multiProcessorCount * sm_per_multiproc * prop.clockRate;
		if (score > maxScore) 
		{
			maxId = id;
			maxScore = score;
		}
	}

	if (maxId < 0) throw "All cuda devices prohibited";

	rc = hipSetDevice(maxId);
	if (hipSuccess != rc) throw "Unable to set cuda device";

	rc = hipGetDeviceProperties(&prop, maxId);
	if (hipSuccess != rc) throw "Unable to get device properties";

	printf("\nSelected GPU %d: \"%s\" with compute capability %d.%d\n\n", 
		maxId, prop.name, prop.major, prop.minor);
}
#if USE_KINECT
class SaveKinectCapture : public AsyncWork
{
public:
	SaveKinectCapture(const char* filename, Kinect::Capture* capture)
	{
		_capture = capture;
		_filename = strdup(filename);
	}
	~SaveKinectCapture()
	{
		free(_filename);
		delete _capture;
	}
	virtual void doWork() override
	{
		printf("Saving color\n");
		
		char filenameWithExt[128];
		sprintf(filenameWithExt, "%s.jpg", _filename);
		File colorFile(_capture->color.data, _capture->color.size, false);
		colorFile.save(filenameWithExt);

		sprintf(filenameWithExt, "%s.d16.lz4", _filename);
		File depthFile(_capture->depth.data, _capture->depth.size, false);
		depthFile.saveCompressed(filenameWithExt);

		printf("DONE\n");
	}

private:
	Kinect::Capture* _capture;
	char* _filename;
};
#endif

int main(int /*argc*/, char** /*argv*/)
{
	int rc;
	hipStream_t stream = 0;

	try 
	{
		printf("Selecting the best GPU\n");
		selectGPU();
		
		rc = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
		if (hipSuccess != rc) throw "Unable to create CUDA stream";

		const char* jpegPath = "sheep.jpg";
		printf("Loading \"%s\"\n", jpegPath);
		
		JpegCodec codec;
		codec.prepare(WIDTH, HEIGHT, 3);
		{
			hipMalloc(&imageBuffer, WIDTH * HEIGHT * 3);
			
			File jpeg;
			jpeg.readAll(jpegPath);
#if USE_NVJPEG
			codec.decodeToDeviceMemoryGPU(imageBuffer, jpeg.data(), jpeg.size(), stream);
#else
			codec.decodeToDeviceMemoryCPU(imageBuffer, jpeg.data(), jpeg.size(), stream);
#endif
			hipStreamSynchronize(stream);
		}
	
		// copy to output folder
		const char* modelPath = "models/fcn_resnet101.960x540.engine";
		printf("Loading \"%s\"", modelPath);
		Model model(modelPath);

		printf("Creating screen\n");
		CudaDisplay display(TITLE, WIDTH, HEIGHT); 
		hipDeviceSynchronize();
		
#if USE_KINECT
		printf("Starting kinect\n");
		Kinect kinect;
		kinect.open();
		kinect.start();
		
		printf("Creating async work queue for kinect capture saving\n");
		AsyncWorkQueue work(4,1000);

		int frame_index = 0;
		char filename[128];
#endif


		dim3 blockSize = { 16, 16 };
		dim3 gridSize = { 
			(WIDTH  + blockSize.x - 1) / blockSize.x, 
			(HEIGHT + blockSize.y - 1) / blockSize.y 
		}; 

		display.cudaMap(stream);
		while (true)
		{
			f_test<<<gridSize, blockSize, 0, stream>>>(
				display.CUDA.frame.data,
				display.CUDA.frame.pitch,
				display.CUDA.frame.width,
				display.CUDA.frame.height
			);

#if USE_KINECT
			auto capture = kinect.capture();

			if (capture)
			{
				printf("jpeg size: %0.03f Kb\n", 0.001 * capture->color.size);
#if USE_NVJPEG
				codec.decodeToDeviceMemoryGPU(
#else
				codec.decodeToDeviceMemoryCPU(
#endif
						imageBuffer, 
						capture->color.data, 
						capture->color.size, 
						stream);
				sprintf(filename, "kinect_%04d", frame_index++);
				hipStreamSynchronize(stream);
				auto savework = new SaveKinectCapture(filename,capture);
				work.enqueue(savework);
			}
#endif
			f_normalize<<<gridSize, blockSize, 0, stream>>>(
				(float*)model.inputFrame.data,
				imageBuffer,
				display.CUDA.frame.width,
				display.CUDA.frame.height
			);
			f_jpeg<<<gridSize, blockSize, 0, stream>>>(
				display.CUDA.frame.data,
				display.CUDA.frame.pitch,
				imageBuffer,
				display.CUDA.frame.width,
				display.CUDA.frame.height
			);
#if 0	
			model.infer(stream);
#endif
			f_segment<<<gridSize, blockSize, 0, stream>>>(
				display.CUDA.frame.data,
				display.CUDA.frame.pitch,
				(int*)model.outputFrame.data,
				display.CUDA.frame.width,
				display.CUDA.frame.height
			);

			// copies the CUDA.frame.data to GL.pbaddr
			// and unmaps the GL.pbo
			display.cudaFinish(stream);
			display.render(stream);
			
			rc = hipGetLastError();
			if (hipSuccess != rc) throw "CUDA ERROR";

			// check escape pressed
			if (display.events()) 
			{
#if USE_KINECT
				kinect.stop();
				kinect.close();
#endif
				display.cudaUnmap(stream);
				hipStreamDestroy(stream);
				return 0;
			}
			usleep(1000);
		}
	}
	catch (const char* &ex)
	{
		fprintf(stderr, "ERROR: %s\n", ex);
		fflush(stderr);
	 	return 1;
	}
	return 0;
}
